#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <stdio.h>

struct square { // older cards could use __mul24 here to do this on the FPU
	__device__ int operator()(int n) {
		return n*n;
	}
};

using namespace thrust;

int main() {
	device_vector<int> d_sequence(100);

	sequence(d_sequence.begin(), d_sequence.end(), 1);
	int squareOfSum = pow(reduce(d_sequence.begin(), d_sequence.end()), 2);

	transform(d_sequence.begin(), d_sequence.end(), d_sequence.begin(), square());
	int sumOfSquares = reduce(d_sequence.begin(), d_sequence.end());

	std::cout << "answer is: " << squareOfSum - sumOfSquares << std::endl;

    return 0;
}

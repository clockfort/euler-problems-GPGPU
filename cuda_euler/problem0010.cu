#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <stdio.h>

const int UPPER_BOUND = 2000000;
__device__ __int64* input_ptr;
__device__ __int64* output_ptr;

struct sieve {
	__device__ __int64 operator()(int index) {
		if (index <= 1) {
			output_ptr[index] = 0;
		}
		else {
			for (__int64 multiplier = 2, result; (result = multiplier * index) < UPPER_BOUND; multiplier++) {
				output_ptr[result] = 0; // zero out all composites
			}
		}
		return index;
	}
};

using namespace thrust;

int main() {
	device_vector<__int64> d_input(UPPER_BOUND);
	device_vector<__int64> d_output(UPPER_BOUND);
	__int64* h_input_ptr = raw_pointer_cast(d_input.data());
	__int64* h_output_ptr = raw_pointer_cast(d_output.data());
	hipMemcpyToSymbol(HIP_SYMBOL(input_ptr), &h_input_ptr, sizeof(__int64*));
	hipMemcpyToSymbol(HIP_SYMBOL(output_ptr), &h_output_ptr, sizeof(__int64*));

	sequence(d_output.begin(), d_output.end());

	transform(make_counting_iterator(0), make_counting_iterator(UPPER_BOUND), d_input.begin(), sieve());

	__int64 sum = reduce(d_output.begin(), d_output.end());
	std::cout << "answer is: " << sum << std::endl;

	return 0;
}

#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform_reduce.h>
#include <stdio.h>

const int UPPER_BOUND = 1000;
__device__ int* square_lut;

struct pythagoreanTripletFilter {
	__device__ int operator()(int a) {
		for (int b = 1; b < a; b++) {
			int apb = square_lut[a] + square_lut[b];
			int c = sqrt((double)apb);
			if (a + b + c == UPPER_BOUND && c * c == apb) {
				return a*b*c;
			}
		}
		return 0;
	}
};

using namespace thrust;

int main() {
	device_vector<int> d_square(UPPER_BOUND);
	device_vector<int> d_sequence(UPPER_BOUND);

	int* h_input_ptr = raw_pointer_cast(d_square.data());
	hipMemcpyToSymbol(HIP_SYMBOL(square_lut), &h_input_ptr, sizeof(int*));

	// construct a LUT for small squares
	sequence(d_square.begin(), d_square.end());
	sequence(d_sequence.begin(), d_sequence.end());
	transform(d_sequence.begin(), d_sequence.end(), d_square.begin(), d_square.begin(), multiplies<__int64>());

	int justAnswer = transform_reduce(d_sequence.begin(), d_sequence.end(), pythagoreanTripletFilter(), 0, maximum<int>());

	std::cout << justAnswer << std::endl;

	return 0;
}
